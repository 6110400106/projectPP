
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <stdbool.h>
#include <limits.h>
#include <time.h>

#define inf 9999
#define NV 5
#define tolerance 0.001

void createGraph(float *arr, int N) {
    time_t t;                               // used for randomizing values
    int col; 
    int row;
    int maxWeight = 100;                    // limit the weight an edge can have

    srand((unsigned) time(&t));             // generate random

    for (col = 0; col < sqrt(N); col++) { 
        for(row = 0; row < sqrt(N); row++) {
            if( col != row){
                arr[(int)(row*sqrt(N)) + col] = rand() % maxWeight; // assign random

                // have a symmetric graph
                arr[(int)(col*sqrt(N)) + row] = arr[(int)(row*sqrt(N)) + col];
            }
            else
                arr[(int)(row*sqrt(N)) + col] = 0; // NO LOOPS
        }
    }
}

void printGraph(float *arr, int n) {
    for (int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            printf("%f   ", arr[i * n + j]);
        }
        printf("\n");
    }
}

__global__ void floyd0(int n, float* x, int* qx) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int j = ix & (n - 1);
    float tmp;
    for(int k = 0; k < n; k++) {
        tmp = x[ix - j + k] + x[k * n + j];
        // D[i * n + j] > (D[i * n + k] + D[k * n + j])
        if(x[ix * n + j] > x[ix * n + k] + x[k * n + j]) {
            x[ix * n + j] = tmp;
            qx[ix * n + j] = k;
        }
        if(x[ix * n + j] == inf) {
            qx[ix * n + j] = k;
        }
        
    }
}

__global__ void floyd(int n, int k, float* x, int* qx) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int j = ix & (n - 1);
    float temp2 = x[ix - j + k] + x[k * n + j];
    if (x[ix] > temp2) {
        x[ix] = temp2;
        qx[ix] = k;
    }
    if (x[ix] == inf) {
        qx[ix] = -2;
    }
}

__global__ void floyd2(int n, int k, float* x, int* qx) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int j = ix & (n - 1);
    float temp2 = x[ix - j + k] + x[k * n + j];
    if (x[ix] > temp2) {
        x[ix] = temp2;
        qx[ix] = k;
    }
}

void cpu_floyd(int n, float* D, int* Q) {
    for (int k = 0; k < n; k++) {
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < n; j++) {
                if (D[i * n + j] > (D[i * n + k] + D[k * n + j])) {
                    D[i * n + j] = D[i * n + k] + D[k * n + j];
                    Q[i * n + j] = k;
                } 
                if (D[i * n + j] == inf) {
                    Q[i*n+j]=-2;
                }
            }
        }
    }
}

void valid(int n, float* D, float* host_D) {
    printf("VALIDATING THAT D array from CPU and host_D array from GPU match... \n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            // if (abs(D[i * n + j] - host_D[i * n + j]) > tolerance) {
            //     printf("ERROR MISMATCH in array D i %d j %d CPU SAYS %f and GPU SAYS %f \n", i, j, D[i * n + j], host_D[i * n + j]);
            // }
            if (D[i * n + j] != host_D[i * n + j]) {
                printf("ERROR MISMATCH in array D i %d j %d CPU SAYS %f and GPU SAYS %f \n", i, j, D[i * n + j], host_D[i * n + j]);
            }
        }
    }
    printf("OK \n");
}

int main(int argc, char **argv) {
    clock_t t;
    float *host_A, *host_D;
    int *host_Q;
    float *dev_x;
    int *dev_qx;
    float *A, *D;
    int *Q;
    
    int i, j, bk;
    //int k = 0;
    int n = NV;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("\n");
    printf("RUNNING WITH %d VERTICES \n", n);
    printf("\n");

    hipMalloc(&dev_x, n * n * sizeof (float));
    hipMalloc(&dev_qx, n * n * sizeof (float));

    //CPU arrays
    A = (float *) malloc(n * n * sizeof (float)); 
    D = (float *) malloc(n * n * sizeof (float)); 
    Q = (int *) malloc(n * n * sizeof (int)); 

    //GPU arrays
    host_A = (float *) malloc(n * n * sizeof (float));
    host_D = (float *) malloc(n * n * sizeof (float));
    host_Q = (int *) malloc(n * n * sizeof (int));

    // Randomize distances in between each node
    createGraph(A, (n*n)); 

    // Printing graph
    printGraph(A, n);

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            Q[i * n + j] = -1;
            D[i * n + j] = A[i * n + j];
            host_A[i * n + j] = A[i * n + j];
            host_Q[i * n + j] = Q[i * n + j];
        }
    }

    // First Mem Copy
    hipMemcpy(dev_x, host_A, n * n * sizeof (float), hipMemcpyHostToDevice);
    hipMemcpy(dev_qx, host_Q, n * n * sizeof (int), hipMemcpyHostToDevice);

    // GPU Calculation
    bk = (int) (n * n / 512);
    int gputhreads = 512;
    if (bk > 0) {
        gputhreads = 512;
    } else {
        bk = 1;
        gputhreads = n*n;
    }
    printf(" \n");
    printf("BLOCKS :   %d      GPU THREADS:     %d \n", bk, gputhreads);
    printf(" \n");

    hipEventRecord(start); 

    // floyd<<<bk, gputhreads>>>(n, k, dev_x, dev_qx);
    // for (k = 1; k < n; k++) 
    //     floyd2<<<bk, gputhreads>>>(n, k, dev_x, dev_qx);
    floyd0<<<bk, gputhreads>>>(n, dev_x, dev_qx);

    hipEventRecord(stop);

    // Second Mem Copy
    hipMemcpy(host_D, dev_x, n * n * sizeof (float), hipMemcpyDeviceToHost);
    hipMemcpy(host_Q, dev_qx, n * n * sizeof (int), hipMemcpyDeviceToHost);
    
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("GPU Calculation Time elapsed: %.20f milliseconds\n", milliseconds);
    printf("\n");
    
    // CPU calculation
    t = clock();
    cpu_floyd(n, D, Q);

    t = clock() - t;
    printf("CPU Calculation Time elapsed: %.20f milliseconds\n", (((float)t)/CLOCKS_PER_SEC));

    // Check validation of D array from CPU calc and host_D array from GPU calc
    // See if the two arrays match
    valid(n, D, host_D);

    hipFree(dev_x);
    hipFree(dev_qx);

    free(A);
    free(D);
    free(Q);
    free(host_A);
    free(host_D);
    free(host_Q);

    printf("ALL OK WE ARE DONE \n");
    return 0;
}