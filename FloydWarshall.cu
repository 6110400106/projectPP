
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <limits.h>
#include <time.h>

#define NV 5                                // number of vertices

void createGraph(float *arr, int N) {
    time_t t;                               // used for randomizing values
    int col; 
    int row;
    int maxWeight = 100;                    // limit the weight an edge can have

    srand((unsigned) time(&t));             // generate random

    for (col = 0; col < sqrt(N); col++) { 
        for(row = 0; row < sqrt(N); row++) {
            if( col != row){
                arr[(int)(row*sqrt(N)) + col] = rand() % maxWeight; // assign random

                // have a symmetric graph
                arr[(int)(col*sqrt(N)) + row] = arr[(int)(row*sqrt(N)) + col];
            }
            else
                arr[(int)(row*sqrt(N)) + col] = 0; // NO LOOPS
        }
    }
}

void printGraph(float *arr, int n) {
    for (int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            printf("%f      ", arr[i * n + j]);
        }
        printf("\n");
    }
}

__global__ void gpuFloyd(int n, float* arr, int k) {
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if(gid >= n) {
        return;
    }
 
    int idx = n * blockIdx.y + gid;
    __shared__  int shortest_distance;

    if(tid == 0) {
        shortest_distance = arr[n * blockIdx.y + k];
    }

    __syncthreads();

    int node_distance = arr[k * n + gid];
    int total_distance = shortest_distance + node_distance;
    if (arr[idx] > total_distance){
       arr[idx] = total_distance;
    }

    __syncthreads();

}

void cpuFloyd(int n, float* cpuGraph) {
    for (int k = 0; k < n; k++) {
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < n; j++) {
                if (cpuGraph[i * n + j] > (cpuGraph[i * n + k] + cpuGraph[k * n + j])) {
                    cpuGraph[i * n + j] = cpuGraph[i * n + k] + cpuGraph[k * n + j];
                } 
            }
        }
    }
}

void valid(int n, float* cpuGraph, float* gpuGraph) {
    printf("VALIDATING that cpuGraph array from CPU and gpuGraph array from GPU match... \n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (cpuGraph[i * n + j] != gpuGraph[i * n + j]) {
                printf("ERROR MISMATCH in array cpuGraph i %d j %d CPU SAYS %f and GPU SAYS %f \n", i, j, cpuGraph[i * n + j], gpuGraph[i * n + j]);
            }
        }
    }
    printf("OK \n\n");
}

int main(int argc, char **argv) {
    clock_t t;
    float *hostArr, *gpuGraph;
    float *devArr;
    float *graph, *cpuGraph;
    
    int i, j;
    int n = NV;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("\n");
    printf("RUNNING WITH %d VERTICES \n", n);
    printf("\n");

    hipMalloc(&devArr, n * n * sizeof (float));

    //CPU arrays
    graph = (float *) malloc(n * n * sizeof (float)); 
    cpuGraph = (float *) malloc(n * n * sizeof (float)); 

    //GPU arrays
    hostArr = (float *) malloc(n * n * sizeof (float));
    gpuGraph = (float *) malloc(n * n * sizeof (float));

    // Randomize distances in between each node
    createGraph(graph, (n*n)); 

    // Printing graph
    //printGraph(graph, n);

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            cpuGraph[i * n + j] = graph[i * n + j];
            hostArr[i * n + j] = graph[i * n + j];
        }
    }

    // First Mem Copy
    hipMemcpy(devArr, hostArr, n * n * sizeof (float), hipMemcpyHostToDevice);

    // For GPU Calculation
    int gputhreads = 512;

    // Kernel call
    // dim3 dimGrid(n, n, 1);
    dim3 dimGrid((n + gputhreads - 1) / gputhreads, n);  
    hipEventRecord(start); 
    for(int k = 0; k < n; k++) {
        gpuFloyd<<<dimGrid, gputhreads>>>(n, devArr, k);
    }
    hipEventRecord(stop);

    // Second Mem Copy
    hipMemcpy(gpuGraph, devArr, n * n * sizeof (float), hipMemcpyDeviceToHost);
    
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("GPU Calculation Time elapsed: %.20f milliseconds\n", milliseconds);
    
    // CPU calculation
    t = clock();
    cpuFloyd(n, cpuGraph);

    t = clock() - t;
    printf("CPU Calculation Time elapsed: %.20f milliseconds\n\n", (((float)t)/CLOCKS_PER_SEC)*1000);

    // Check validation of cpuGraph array from CPU calc and gpuGraph array from GPU calc
    // See if the two arrays match
    valid(n, cpuGraph, gpuGraph);

    /*
    printf("Graph from GPU:\n");
    printGraph(gpuGraph, n);
    printf("\n");

    printf("Graph from CPU:\n");
    printGraph(cpuGraph, n);
    printf("\n");
    */

    hipFree(devArr);

    free(graph);
    free(cpuGraph);
    free(hostArr);
    free(gpuGraph);

    printf("FINISHED!!! \n");
    return 0;
}